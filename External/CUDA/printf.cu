#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
__global__ void kernel(char a, short b, int c, float d, double e) {
  const char* f = "abcdefg";
  printf("kernel_printf %c %d %d %f %f %s\n", a, b, c, d, e, f);
}

int main(int argc, char* argv[]) {
  hipLaunchKernelGGL((kernel), dim3(2), dim3(8), 0, 0, 'a', 2, 3, 4, 5);
  hipDeviceSynchronize();
  hipDeviceReset();
  return 0;
}
