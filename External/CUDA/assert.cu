#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>

// Use fixed value for __FILE__ so assert message does not depend on
// the actual path to the file during compilation..
#define __FILE__ "assert.cu"

__global__ void kernel() {
  // Our reference output contains the line number of this assert() call; be
  // careful when modifying the parts of this file above this line.
  assert(false);
}

int main() {
  hipLaunchKernelGGL((kernel), dim3(1), dim3(1), 0, 0, );
  hipError_t err = hipDeviceSynchronize();
  if (err != hipErrorAssert)
    return err;
  return 0;
}
