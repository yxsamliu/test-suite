#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <iostream>

__global__ void axpy(float a, float* x, float* y) {
  y[hipThreadIdx_x] = a * x[hipThreadIdx_x];
}

int main(int argc, char* argv[]) {
  const int kDataLen = 4;

  float a = 2.0f;
  float host_x[kDataLen] = {1.0f, 2.0f, 3.0f, 4.0f};
  float host_y[kDataLen];

  // Copy input data to device.
  float* device_x;
  float* device_y;
  hipMalloc(&device_x, kDataLen * sizeof(float));
  hipMalloc(&device_y, kDataLen * sizeof(float));
  hipMemcpy(device_x, host_x, kDataLen * sizeof(float),
             hipMemcpyHostToDevice);

  // Launch the kernel.
  hipLaunchKernelGGL((axpy), dim3(1), dim3(kDataLen), 0, 0, a, device_x, device_y);

  // Copy output data to host.
  hipDeviceSynchronize();
  hipMemcpy(host_y, device_y, kDataLen * sizeof(float),
             hipMemcpyDeviceToHost);

  // Print the results.
  for (int i = 0; i < kDataLen; ++i) {
    std::cout << "y[" << i << "] = " << host_y[i] << "\n";
  }

  hipDeviceReset();
  return 0;
}
